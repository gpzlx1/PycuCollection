#include "hip/hip_runtime.h"
#include <thrust/iterator/zip_iterator.h>
#include <cuco/static_map.cuh>
#include <iostream>

#include "common.h"
#include "cuco_hashmap.h"

template <typename Key, typename Value>
class CUCOHashmap : public Hashmap {
 public:
  using map_type = cuco::static_map<
      Key, Value, std::size_t, cuda::thread_scope_device, thrust::equal_to<Key>,
      cuco::linear_probing<4, cuco::default_hash_function<Key>>,
      cuco::cuda_allocator<cuco::pair<Key, Value>>, cuco::storage<1>>;

  CUCOHashmap(torch::Tensor keys, torch::Tensor values, double load_factor) {
    Key constexpr empty_key_sentinel = -1;
    Value constexpr empty_value_sentinel = -1;

    int64_t numel = keys.numel();
    std::size_t const capacity = std::ceil(numel / load_factor);

    // Create a cuco::static_map
    map_ = new map_type(capacity, cuco::empty_key{empty_key_sentinel},
                        cuco::empty_value{empty_value_sentinel});
    auto zipped = thrust::make_zip_iterator(
        thrust::make_tuple(keys.data_ptr<Key>(), values.data_ptr<Value>()));
    map_->insert(zipped, zipped + numel);

    // Set property
    key_options_ = keys.options();
    value_options_ = values.options();
    capacity_ = capacity;
    memory_usage_ = 100;  // for test
  };

  ~CUCOHashmap() { delete map_; };

  torch::Tensor query(torch::Tensor requests) {
    int64_t numel = requests.numel();
    torch::Tensor result = torch::full_like(requests, -1, value_options_);
    map_->find(requests.data_ptr<Key>(), requests.data_ptr<Key>() + numel,
               result.data_ptr<Value>());
    return result;
  };

 private:
  torch::TensorOptions key_options_;
  torch::TensorOptions value_options_;
  // int64_t memory_usage_;
  // int64_t capacity_;
  map_type* map_;
};

CUCOHashmapWrapper::CUCOHashmapWrapper(torch::Tensor keys, torch::Tensor values,
                                       double load_factor) {
  CHECK_CUDA(keys);
  CHECK_CUDA(values);
  key_type_ = keys.dtype();
  value_type_ = values.dtype();

  INTEGER_TYPE_SWITCH(key_type_, Key, {
    INTEGER_TYPE_SWITCH(value_type_, Value, {
      map_ = new CUCOHashmap<Key, Value>(keys, values, load_factor);
    });
  });
}

torch::Tensor CUCOHashmapWrapper::query(torch::Tensor requests) {
  CHECK_CUDA(requests);
  INTEGER_TYPE_SWITCH(key_type_, Key, {
    INTEGER_TYPE_SWITCH(value_type_, Value, {
      auto map = (CUCOHashmap<Key, Value>*)map_;
      return map->query(requests.to(key_type_));
    });
  });

  return torch::Tensor();
}
